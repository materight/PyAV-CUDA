#include <stdint.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 32

template<class T>
__device__ static T clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}


template<bool FullColorRange>
__global__ void nv12_to_rgb_kernel(
    uint8_t *inY,
    uint8_t *inUV,
    uint8_t *outRGB,
    int height,
    int width,
    int pitch
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int yIdx = y * pitch + x;
    int uvIdx = (y / 2) * pitch + (x / 2) * 2;

    uint8_t Y = inY[yIdx];
    uint8_t U = inUV[uvIdx];
    uint8_t V = inUV[uvIdx + 1];

    float fY = (int)Y - 0;
    float fU = (int)U - 128;
    float fV = (int)V - 128;

    uint8_t R, G, B;
    if constexpr (FullColorRange) {
        R = clamp(1.000f * fY +             + 1.402f * fV, 0.0f, 255.0f);
        G = clamp(1.000f * fY - 0.344f * fU - 0.714f * fV, 0.0f, 255.0f);
        B = clamp(1.000f * fY + 1.772f * fU              , 0.0f, 255.0f);
    } else {
        fY -= 16;
        R = clamp(1.164f * fY +             + 1.596f * fV, 0.0f, 255.0f);
        G = clamp(1.164f * fY - 0.392f * fU - 0.813f * fV, 0.0f, 255.0f);
        B = clamp(1.164f * fY + 2.017f * fU              , 0.0f, 255.0f);
    }

    int rgbIdx = (y * width + x) * 3;
    outRGB[rgbIdx] = R;
    outRGB[rgbIdx + 1] = G;
    outRGB[rgbIdx + 2] = B;
}


__global__ void rgb_to_nv12_kernel(
    uint8_t *inRGB,
    uint8_t *outY,
    uint8_t *outUV,
    int height,
    int width,
    int pitch
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int rgbIdx = (y * width + x) * 3;
    uint8_t R = inRGB[rgbIdx];
    uint8_t G = inRGB[rgbIdx + 1];
    uint8_t B = inRGB[rgbIdx + 2];

    uint8_t Y = clamp(0.257f * R + 0.504f * G + 0.098f * B + 16.0f, 0.0f, 255.0f);

    int yIdx = y * pitch + x;
    outY[yIdx] = Y;

    if ((x % 2 == 0) && (y % 2 == 0)) {
        uint8_t U = clamp(-0.148f * R - 0.291f * G + 0.439f * B + 128.0f, 0.0f, 255.0f);
        uint8_t V = clamp( 0.439f * R - 0.368f * G - 0.071f * B + 128.0f, 0.0f, 255.0f);

        int uvIdx = (y / 2) * pitch + (x / 2) * 2;
        outUV[uvIdx] = U;
        outUV[uvIdx + 1] = V;
    }
}


inline int divCeil(int num, int den) {
    return (num + (den - 1)) / den;
}

hipError_t checkCudaErrorAndSync() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) return err;
    err = hipDeviceSynchronize();
    if (err != hipSuccess) return err;
    return hipSuccess;
}


extern "C" {
    hipError_t NV12ToRGB(uint8_t *inY, uint8_t *inUV, uint8_t *outRGB, int height, int width, int pitch, bool fullColorRange) {
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
        dim3 gridSize(divCeil(width, blockSize.x), divCeil(height, blockSize.y));

        if (fullColorRange) {
            nv12_to_rgb_kernel<true><<<gridSize, blockSize>>>(inY, inUV, outRGB, height, width, pitch);
        } else {
            nv12_to_rgb_kernel<false><<<gridSize, blockSize>>>(inY, inUV, outRGB, height, width, pitch);
        }

        return checkCudaErrorAndSync();
    }

    hipError_t RGBToNV12(uint8_t *inRGB, uint8_t *outY, uint8_t *outUV, int height, int width, int pitch) {
        dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE / 4);
        dim3 gridSize(divCeil(width, blockSize.x), divCeil(height, blockSize.y));

        rgb_to_nv12_kernel<<<gridSize, blockSize>>>(inRGB, outY, outUV, height, width, pitch);

        return checkCudaErrorAndSync();
    }
}